#include "hip/hip_runtime.h"
﻿// kernel_parallel.cu
#include <iostream>
#include <cstdlib>
#include <chrono>
#include <hip/hip_runtime.h>

#define N 100000 // Tamanho da matriz/vetor

__global__ void multiplicacaoMatrizVetorKernel(float* matriz, float* vetor, float* resultado, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        float soma = 0.0;
        for (int j = 0; j < n; ++j) {
            soma += matriz[i * n + j] * vetor[j];
        }
        resultado[i] = soma;
    }
}

int main() {
    int n = N;
    size_t bytes = n * n * sizeof(float);
    size_t bytesVetor = n * sizeof(float);

    float* h_matriz = (float*)malloc(bytes);
    float* h_vetor = (float*)malloc(bytesVetor);
    float* h_resultado = (float*)malloc(bytesVetor);

    // Inicialização aleatória da matriz e vetor
    for (int i = 0; i < n; ++i) {
        h_vetor[i] = rand() % 100;
        for (int j = 0; j < n; ++j) {
            h_matriz[i * n + j] = rand() % 100;
        }
    }

    float* d_matriz, * d_vetor, * d_resultado;
    hipMalloc(&d_matriz, bytes);
    hipMalloc(&d_vetor, bytesVetor);
    hipMalloc(&d_resultado, bytesVetor);

    hipMemcpy(d_matriz, h_matriz, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_vetor, h_vetor, bytesVetor, hipMemcpyHostToDevice);

    int blockSize = 100000; // Pode ser alterado para 512 se desejado
    int gridSize = (n + blockSize - 1) / blockSize; // Número de blocos

    // Teste de tempo
    auto inicio = std::chrono::high_resolution_clock::now();
    multiplicacaoMatrizVetorKernel << <gridSize, blockSize >> > (d_matriz, d_vetor, d_resultado, n);
    hipDeviceSynchronize();
    auto fim = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> duracao = fim - inicio;

    // Copiar resultado para a memória do host (sem exibir)
    hipMemcpy(h_resultado, d_resultado, bytesVetor, hipMemcpyDeviceToHost);

    std::cout << "Tempo de execução (Paralelo): " << duracao.count() << " segundos\n";

    // Liberar memória
    hipFree(d_matriz);
    hipFree(d_vetor);
    hipFree(d_resultado);
    free(h_matriz);
    free(h_vetor);
    free(h_resultado);

    return 0;
}
