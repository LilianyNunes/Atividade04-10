﻿// kernel_serial.cu

#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <chrono>

#define N 100000 // Tamanho da matriz/vetor

void multiplicacaoMatrizVetor(float* matriz, float* vetor, float* resultado, int n) {
    for (int i = 0; i < n; ++i) {
        float soma = 0.0;
        for (int j = 0; j < n; ++j) {
            soma += matriz[i * n + j] * vetor[j];
        }
        resultado[i] = soma;
    }
}

int main() {
    int n = N;
    size_t bytes = n * n * sizeof(float);
    size_t bytesVetor = n * sizeof(float);

    float* h_matriz = (float*)malloc(bytes);
    float* h_vetor = (float*)malloc(bytesVetor);
    float* h_resultado = (float*)malloc(bytesVetor);

    // Inicialização aleatória da matriz e vetor
    for (int i = 0; i < n; ++i) {
        h_vetor[i] = rand() % 100;
        for (int j = 0; j < n; ++j) {
            h_matriz[i * n + j] = rand() % 100;
        }
    }

    // Teste de tempo
    auto inicio = std::chrono::high_resolution_clock::now();
    multiplicacaoMatrizVetor(h_matriz, h_vetor, h_resultado, n);
    auto fim = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> duracao = fim - inicio;

    std::cout << "Tempo de execução (Serial): " << duracao.count() << " segundos\n";

    free(h_matriz);
    free(h_vetor);
    free(h_resultado);

    return 0;
}
